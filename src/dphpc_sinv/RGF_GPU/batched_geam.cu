#include "hip/hip_runtime.h"
#include "batched_geam.h"

__global__ void quatrexblasZgeamBatched_C_C_kernel(
    int m, int n,
    hipDoubleComplex alpha,
    hipDoubleComplex *const * __restrict__ Aarray, int lda,
    hipDoubleComplex beta,
    hipDoubleComplex *const * __restrict__ Barray, int ldb,
    hipDoubleComplex ** __restrict__ Carray, int ldc,
    int batchSize
){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int stride = blockDim.x*gridDim.x;

    for(int id = tid; id < m*n*batchSize; id += stride){
        int batch = id/(m*n);
        int ij = id % (m*n);
        int row = ij % m;
        int col = ij / m;

        hipDoubleComplex a = hipConj(Aarray[batch][col + row*lda]);
        hipDoubleComplex b = hipConj(Barray[batch][col + row*ldb]);

        Carray[batch][row + col*ldc] = hipCadd(hipCmul(alpha, a), hipCmul(beta, b));
    }
}

__global__ void quatrexblasZgeamBatched_N_C_kernel(
    int m, int n,
    hipDoubleComplex alpha,
    hipDoubleComplex *const *Aarray, int lda,
    hipDoubleComplex beta,
    hipDoubleComplex *const * __restrict__ Barray, int ldb,
    hipDoubleComplex **Carray, int ldc,
    int batchSize
){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int stride = blockDim.x*gridDim.x;

    for(int id = tid; id < m*n*batchSize; id += stride){
        int batch = id/(m*n);
        int ij = id % (m*n);
        int row = ij % m;
        int col = ij / m;

        hipDoubleComplex a = Aarray[batch][row + col*ldc];
        hipDoubleComplex b = hipConj(Barray[batch][col + row*ldb]);

        Carray[batch][row + col*ldc] = hipCadd(hipCmul(alpha, a), hipCmul(beta, b));
    }
}

void quatrexblasZgeamBatched_C_C(
    hipblasHandle_t handle,
    int m, int n,
    hipDoubleComplex *alpha,
    hipDoubleComplex *const Aarray[], int lda,
    hipDoubleComplex *beta,
    hipDoubleComplex *const Barray[], int ldb,
    hipDoubleComplex *Carray[], int ldc,
    int batchSize
){
    hipStream_t stream;
    cublasErrchk(hipblasGetStream(handle, &stream));

    int threads = 1024;
    int blocks = (m*n*batchSize + threads - 1)/threads;
    quatrexblasZgeamBatched_C_C_kernel<<<blocks, threads, 0, stream>>>(
        m, n,
        alpha[0],
        Aarray, lda,
        beta[0],
        Barray, ldb,
        Carray, ldc,
        batchSize
    );
}

void quatrexblasZgeamBatched_N_C(
    hipblasHandle_t handle,
    int m, int n,
    hipDoubleComplex *alpha,
    hipDoubleComplex *const Aarray[], int lda,
    hipDoubleComplex *beta,
    hipDoubleComplex *const Barray[], int ldb,
    hipDoubleComplex *Carray[], int ldc,
    int batchSize
){
    hipStream_t stream;
    cublasErrchk(hipblasGetStream(handle, &stream));

    int threads = 1024;
    int blocks = (m*n*batchSize + threads - 1)/threads;
    quatrexblasZgeamBatched_N_C_kernel<<<blocks, threads, 0, stream>>>(
        m, n,
        alpha[0],
        Aarray, lda,
        beta[0],
        Barray, ldb,
        Carray, ldc,
        batchSize
    );
}

hipblasStatus_t quatrexblasZgeamBatched(
    hipblasHandle_t handle,
    hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n,
    hipDoubleComplex *alpha,
    hipDoubleComplex *const Aarray[], int lda,
    hipDoubleComplex *beta,
    hipDoubleComplex *const Barray[], int ldb,
    hipDoubleComplex *Carray[], int ldc,
    int batchSize
){
    // assumes arrays are column major
    //TODO: Add error checking
    // inputs are not checked for validity

    if(transa == HIPBLAS_OP_C && transb == HIPBLAS_OP_C){
        quatrexblasZgeamBatched_C_C(
            handle,
            m, n,
            alpha,
            Aarray, lda,
            beta,
            Barray, ldb,
            Carray, ldc,
            batchSize
        );
    }
    else if(transa == HIPBLAS_OP_N && transb == HIPBLAS_OP_C){
        quatrexblasZgeamBatched_N_C(
            handle,
            m, n,
            alpha,
            Aarray, lda,
            beta,
            Barray, ldb,
            Carray, ldc,
            batchSize
        );
    }
    else{
        return HIPBLAS_STATUS_NOT_SUPPORTED;
    }

    return HIPBLAS_STATUS_SUCCESS;

}