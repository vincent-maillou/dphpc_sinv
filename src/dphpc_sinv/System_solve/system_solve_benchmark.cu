#include "hip/hip_runtime.h"
#include <string> 
#include <omp.h>

#include "utils.h"

#include "mkl.h"
#include "hipsolver.h"
#include <hipsparse.h>



// cusolver has HIPSOLVER_STATUS_SUCCESS and not hipSuccess, but they are the same
// this seems again kinda hacky
#define cudaErrchk(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDAassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define cusolverErrchk(ans) { cusolverAssert((ans), __FILE__, __LINE__); }
inline void cusolverAssert(hipsolverStatus_t code, const char *file, int line, bool abort=true)
{
   if (code != HIPSOLVER_STATUS_SUCCESS) 
   {
        //Did not find a counter part to hipGetErrorString in cusolver
        fprintf(stderr,"CUSOLVERassert: %s %d\n", file, line);
        if (abort) exit(code);
   }
}


#define cublasErrchk(ans) { cublasAssert((ans), __FILE__, __LINE__); }
inline void cublasAssert(hipblasStatus_t code, const char *file, int line, bool abort=true)
{
   if (code != HIPBLAS_STATUS_SUCCESS) 
   {
        //Did not find a counter part to hipGetErrorString in cusolver
        fprintf(stderr,"CUBLASassert: %s %d\n", file, line);
        if (abort) exit(code);
   }
}

#define cusparseErrchk(ans) { cusparseAssert((ans), __FILE__, __LINE__); }
inline void cusparseAssert(hipsparseStatus_t code, const char *file, int line, bool abort=true)
{
   if (code != HIPSPARSE_STATUS_SUCCESS) 
   {
        //Did not find a counter part to hipGetErrorString in cusolver
        fprintf(stderr,"CUSPARSEassert: %s %d\n", file, line);
        if (abort) exit(code);
   }
}

hipsolverHandle_t CreateCusolverDnHandle(int device) {
    if (hipSetDevice(device) != hipSuccess) {
        throw std::runtime_error("Failed to set CUDA device.");
    }
    hipsolverHandle_t handle;
    cusolverErrchk(hipsolverDnCreate(&handle));
    return handle;
}


double solve_mkl_dgesv(
    double *matrix_dense,
    double *rhs,
    double *reference_solution,
    int matrice_size,
    double tolerance,
    bool flag_verbose)
{

    double time = -1.0;

    if(flag_verbose){
        std::printf("Copy data to device\n");
    }

    int ipiv[matrice_size];
    int nrhs = 1;
    int info;
    time = -omp_get_wtime();
    dgesv(&matrice_size, &nrhs, matrix_dense, &matrice_size, ipiv, rhs, &matrice_size, &info);
    time += omp_get_wtime();

    if(info != 0){
        std::printf("Error in MKL dgesv\n");
        std::printf("info: %d\n", info);
        return -1.0;
    }

    if(flag_verbose){
        std::printf("MKL dgesv done\n");
    }

    if(!assert_same_array<double>(rhs, reference_solution, tolerance, matrice_size)){
        std::printf("Error: MKL dgesv solution is not the same as the reference solution\n");
        return -1.0;
    }
    else{
        std::printf("MKL dgesv solution is the same as the reference solution\n");
    }
    return time;
}

double solve_cusolver_LU(
    double *matrix_dense_h,
    double *rhs_h,
    double *reference_solution_h,
    int matrice_size,
    double tolerance,
    bool flag_verbose)
{

    double time = -1.0;
    hipStream_t stream = NULL;
    hipsolverHandle_t handle = CreateCusolverDnHandle(0);
    cudaErrchk(hipStreamCreate(&stream));
    cusolverErrchk(hipsolverSetStream(handle, stream));



    int info_h = 0;
    int bufferSize = 0;

    double *matrix_dense_d = NULL;
    double *rhs_d = NULL;
    int *ipiv_d = NULL;
    int *info_d = NULL;
    double *buffer = NULL;

    //allocate memory on device
    cudaErrchk(hipMalloc((void**)&info_d, sizeof(int)))
    cudaErrchk(hipMalloc((void**)&matrix_dense_d, matrice_size*matrice_size*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&rhs_d, matrice_size*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&ipiv_d, matrice_size*sizeof(int)));


    //copy data to device
    if(flag_verbose){
        std::printf("Copy data to device\n");
    }
    cudaErrchk(hipMemcpy(matrix_dense_d, matrix_dense_h, matrice_size*matrice_size*sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemset(info_d, 0, sizeof(int)));
    cudaErrchk(hipMemcpy(rhs_d, rhs_h, matrice_size*sizeof(double), hipMemcpyHostToDevice));


    //figure out extra amount of memory needed
    cusolverErrchk(hipsolverDnDgetrf_bufferSize(handle, matrice_size, matrice_size,
                                            (double *)matrix_dense_d,
                                              matrice_size, &bufferSize));
    cudaErrchk(hipMalloc(&buffer, sizeof(double) * bufferSize));

    //LU factorization
    if(flag_verbose){
        std::printf("LU factorization\n");
    }
    time = -omp_get_wtime();
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipStreamSynchronize(stream));
    cusolverErrchk(hipsolverDnDgetrf(handle, matrice_size, matrice_size,
                                matrix_dense_d, matrice_size, buffer, ipiv_d, info_d));
    
    //copy info to host
    cudaErrchk(hipMemcpy(&info_h, info_d, sizeof(int), hipMemcpyDeviceToHost));

    if (info_h != 0) {
        fprintf(stderr, "Error: LU factorization failed\n");
    }
    else{
        std::printf("LU factorization done\n");
    }

    if(flag_verbose){
        std::printf("Back substitution\n");
    }
    //back substitution
    cusolverErrchk(hipsolverDnDgetrs(handle, HIPBLAS_OP_N, matrice_size,
                                    1, matrix_dense_d, matrice_size, ipiv_d,
                                    rhs_d, matrice_size, info_d));
    cudaErrchk(hipStreamSynchronize(stream));
    cudaErrchk(hipDeviceSynchronize());
    time += omp_get_wtime();


    cudaErrchk(hipMemcpy(&info_h, info_d, sizeof(int), hipMemcpyDeviceToHost));
    if (info_h != 0) {
        fprintf(stderr, "Error: Back substitution failed\n");
    }
    else{
        std::printf("Back substitution done\n");
    }

    //copy solution to host
    if(flag_verbose){
        std::printf("Copy solution to host\n");
    }
    cudaErrchk(hipMemcpy(rhs_h, rhs_d, matrice_size*sizeof(double), hipMemcpyDeviceToHost));

    if(!assert_same_array<double>(rhs_h, reference_solution_h, tolerance, matrice_size)){
        std::printf("Error: CuSolver solution is not the same as the reference solution\n");
        return -1.0;
    }
    else{
        std::printf("CuSolver solution is the same as the reference solution\n");
    }


    if (info_d) {
        cudaErrchk(hipFree(info_d));
    }
    if (buffer) {
        cudaErrchk(hipFree(buffer));
    }
    if (matrix_dense_d) {
        cudaErrchk(hipFree(matrix_dense_d));
    }
    if (ipiv_d) {
        cudaErrchk(hipFree(ipiv_d));
    }


    if (handle) {
        cusolverErrchk(hipsolverDnDestroy(handle));
    }
    if (stream) {
        cudaErrchk(hipStreamDestroy(stream));
    }

    return time;
}


double solve_cusparse_ILU_CG(
    double *data_h,
    int *col_indices_h,
    int *row_indptr_h,
    double *rhs_h,
    double *reference_solution_h,
    int nnz,
    int matrice_size,
    double tolerance,
    bool flag_verbose){

    double time = -1.0;
    hipStream_t stream = NULL;
    
    
    hipsparseHandle_t cusparseHandle = 0;
    cusparseErrchk(hipsparseCreate(&cusparseHandle));    

    hipblasHandle_t cublasHandle = 0;
    cublasErrchk(hipblasCreate(&cublasHandle));

    cudaErrchk(hipStreamCreate(&stream));
    cusparseErrchk(hipsparseSetStream(cusparseHandle, stream));
    cublasErrchk(hipblasSetStream(cublasHandle, stream));


    double *data_d = NULL;
    double *col_indices_d = NULL;
    double *row_indptr_d = NULL;
    double *rhs_d = NULL;
    double *x_d = NULL;
    double *p_d = NULL;
    double *Ax_d = NULL;
    double dot;

    hipsparseSpMatDescr_t matA = NULL;

    const double tol = 1.e-15;
    const int max_iter = 2000;
    double a, b, na;
    double alpha, beta, alpham1, r0, r1;
    size_t bufferSize = 0;
    void *buffer = NULL;

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.0;


    //allocate memory on device
    cudaErrchk(hipMalloc((void**)&data_d, nnz*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&col_indices_d, nnz*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&row_indptr_d, (matrice_size+1)*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&rhs_d, matrice_size*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&x_d, matrice_size*sizeof(double)));
    cudaErrchk(hipMalloc((void **)&p_d, matrice_size * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&Ax_d, matrice_size * sizeof(double)));

    /* Wrap raw data into cuSPARSE generic API objects */
    cusparseErrchk(hipsparseCreateCsr(&matA, matrice_size, matrice_size,
                                        nnz, row_indptr_d, col_indices_d, data_d,
                                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));



    hipsparseDnVecDescr_t vecx = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecx, matrice_size, x_d, HIP_R_64F));
    hipsparseDnVecDescr_t vecp = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecp, matrice_size, p_d, HIP_R_64F));
    hipsparseDnVecDescr_t vecAx = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecAx, matrice_size, Ax_d, HIP_R_64F));


    //copy data to device
    if(flag_verbose){
        std::printf("Copy data to device\n");
    }
    cudaErrchk(hipMemcpy(rhs_d, rhs_h, matrice_size*sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(col_indices_d, col_indices_h, nnz * sizeof(int), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(row_indptr_d, row_indptr_h, (matrice_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(data_d, data_h, nnz * sizeof(double), hipMemcpyHostToDevice));
    
    // setting starting guess to zero
    cudaErrchk(hipMemset(x_d, 0.0, matrice_size*sizeof(double)))
    

    //figure out extra amount of memory needed
    if(flag_verbose){
        std::printf("Figure out extra amount of memory needed\n");
    }
    cusparseErrchk(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecx,
        &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    cudaErrchk(hipMalloc(&buffer, bufferSize));


    //begin CG
    time = -omp_get_wtime();
    cudaErrchk(hipStreamSynchronize(stream));
    cudaErrchk(hipDeviceSynchronize());
    if(flag_verbose){
        std::printf("CG starts\n");
    }

    // calc A*x
    cusparseErrchk(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, vecx, &beta, vecAx, HIP_R_64F,
                               HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

    // r = b - A*x
    cublasErrchk(hipblasDaxpy(cublasHandle, matrice_size, &alpham1, Ax_d, 1, rhs_d, 1));
    cublasErrchk(hipblasDdot(cublasHandle, matrice_size, rhs_d, 1, rhs_d, 1, &r1));


    int k = 1;
    while (r1 > tol * tol && k <= max_iter) {
        if(k > 1){
            b = r1 / r0;
            cublasErrchk(hipblasDscal(cublasHandle, matrice_size, &b, p_d, 1));
            cublasErrchk(hipblasDaxpy(cublasHandle, matrice_size, &alpha, rhs_d, 1, p_d, 1));            
        }
        else {
            cublasErrchk(hipblasDcopy(cublasHandle, matrice_size, rhs_d, 1, p_d, 1));
        }

        cusparseErrchk(hipsparseSpMV(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecp,
            &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
    
        cublasErrchk(hipblasDdot(cublasHandle, matrice_size, p_d, 1, Ax_d, 1, &dot));
        a = r1 / dot;

        cublasErrchk(hipblasDaxpy(cublasHandle, matrice_size, &a, p_d, 1, x_d, 1));
        na = -a;
        cublasErrchk(hipblasDaxpy(cublasHandle, matrice_size, &na, Ax_d, 1, rhs_d, 1));

        r0 = r1;
        cublasErrchk(hipblasDdot(cublasHandle, matrice_size, rhs_d, 1, rhs_d, 1, &r1));
        cudaErrchk(hipStreamSynchronize(stream));
        if(flag_verbose){
            std::printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        }
        k++;
    }

    //end CG
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipStreamSynchronize(stream));
    time += omp_get_wtime();

    //copy solution to host
    if(flag_verbose){
        std::printf("Copy solution to host\n");
    }
    cudaErrchk(hipMemcpy(rhs_h, x_d, matrice_size * sizeof(double), hipMemcpyDeviceToHost));


    if(!assert_same_array<double>(rhs_h, reference_solution_h, tolerance, matrice_size)){
        std::printf("Error: CG solution is not the same as the reference solution\n");
        return -1.0;
    }
    else{
        std::printf("CG solution is the same as the reference solution\n");
    }


    if (buffer) {
        cudaErrchk(hipFree(buffer));
    }
    if(cusparseHandle) {
        cusparseErrchk(hipsparseDestroy(cusparseHandle));
    }
    if(cublasHandle) {
        cublasErrchk(hipblasDestroy(cublasHandle));
    }
    if(stream) {
        cudaErrchk(hipStreamDestroy(stream));
    }
    if(matA) {
        cusparseErrchk(hipsparseDestroySpMat(matA));
    }
    if(vecx) {
        cusparseErrchk(hipsparseDestroyDnVec(vecx));
    }
    if(vecAx) {
        cusparseErrchk(hipsparseDestroyDnVec(vecAx));
    }
    if(vecp) {
        cusparseErrchk(hipsparseDestroyDnVec(vecp));
    }

    if(data_d){
        cudaErrchk(hipFree(data_d));
    }
    if(col_indices_d){
        cudaErrchk(hipFree(col_indices_d));
    }
    if(row_indptr_d){
        cudaErrchk(hipFree(row_indptr_d));
    }
    if(rhs_d){
        cudaErrchk(hipFree(rhs_d));
    }
    if(x_d){
        cudaErrchk(hipFree(x_d));
    }
    if(p_d){
        cudaErrchk(hipFree(p_d));
    }
    if(Ax_d){
        cudaErrchk(hipFree(Ax_d));
    }

    return time;
}

double solve_cusparse_CG(
    double *data_h,
    int *col_indices_h,
    int *row_indptr_h,
    double *rhs_h,
    double *reference_solution_h,
    int nnz,
    int matrice_size,
    double tolerance,
    bool flag_verbose){

    double time = -1.0;
    hipStream_t stream = NULL;
    
    
    hipsparseHandle_t cusparseHandle = 0;
    cusparseErrchk(hipsparseCreate(&cusparseHandle));    

    hipblasHandle_t cublasHandle = 0;
    cublasErrchk(hipblasCreate(&cublasHandle));

    cudaErrchk(hipStreamCreate(&stream));
    cusparseErrchk(hipsparseSetStream(cusparseHandle, stream));
    cublasErrchk(hipblasSetStream(cublasHandle, stream));


    double *data_d = NULL;
    double *col_indices_d = NULL;
    double *row_indptr_d = NULL;
    double *rhs_d = NULL;
    double *x_d = NULL;
    double *p_d = NULL;
    double *Ax_d = NULL;
    double dot;

    hipsparseSpMatDescr_t matA = NULL;

    const double tol = 1.e-15;
    const int max_iter = 2000;
    double a, b, na;
    double alpha, beta, alpham1, r0, r1;
    size_t bufferSize = 0;
    void *buffer = NULL;

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.0;


    //allocate memory on device
    cudaErrchk(hipMalloc((void**)&data_d, nnz*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&col_indices_d, nnz*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&row_indptr_d, (matrice_size+1)*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&rhs_d, matrice_size*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&x_d, matrice_size*sizeof(double)));
    cudaErrchk(hipMalloc((void **)&p_d, matrice_size * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&Ax_d, matrice_size * sizeof(double)));

    /* Wrap raw data into cuSPARSE generic API objects */
    cusparseErrchk(hipsparseCreateCsr(&matA, matrice_size, matrice_size,
                                        nnz, row_indptr_d, col_indices_d, data_d,
                                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));



    hipsparseDnVecDescr_t vecx = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecx, matrice_size, x_d, HIP_R_64F));
    hipsparseDnVecDescr_t vecp = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecp, matrice_size, p_d, HIP_R_64F));
    hipsparseDnVecDescr_t vecAx = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecAx, matrice_size, Ax_d, HIP_R_64F));


    //copy data to device
    if(flag_verbose){
        std::printf("Copy data to device\n");
    }
    cudaErrchk(hipMemcpy(rhs_d, rhs_h, matrice_size*sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(col_indices_d, col_indices_h, nnz * sizeof(int), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(row_indptr_d, row_indptr_h, (matrice_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(data_d, data_h, nnz * sizeof(double), hipMemcpyHostToDevice));
    
    // setting starting guess to zero
    cudaErrchk(hipMemset(x_d, 0.0, matrice_size*sizeof(double)))
    

    //figure out extra amount of memory needed
    if(flag_verbose){
        std::printf("Figure out extra amount of memory needed\n");
    }
    cusparseErrchk(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecx,
        &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    cudaErrchk(hipMalloc(&buffer, bufferSize));


    //begin CG
    time = -omp_get_wtime();
    cudaErrchk(hipStreamSynchronize(stream));
    cudaErrchk(hipDeviceSynchronize());
    if(flag_verbose){
        std::printf("CG starts\n");
    }

    // calc A*x
    cusparseErrchk(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, vecx, &beta, vecAx, HIP_R_64F,
                               HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

    // r = b - A*x
    cublasErrchk(hipblasDaxpy(cublasHandle, matrice_size, &alpham1, Ax_d, 1, rhs_d, 1));
    cublasErrchk(hipblasDdot(cublasHandle, matrice_size, rhs_d, 1, rhs_d, 1, &r1));


    int k = 1;
    while (r1 > tol * tol && k <= max_iter) {
        if(k > 1){
            b = r1 / r0;
            cublasErrchk(hipblasDscal(cublasHandle, matrice_size, &b, p_d, 1));
            cublasErrchk(hipblasDaxpy(cublasHandle, matrice_size, &alpha, rhs_d, 1, p_d, 1));            
        }
        else {
            cublasErrchk(hipblasDcopy(cublasHandle, matrice_size, rhs_d, 1, p_d, 1));
        }

        cusparseErrchk(hipsparseSpMV(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecp,
            &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
    
        cublasErrchk(hipblasDdot(cublasHandle, matrice_size, p_d, 1, Ax_d, 1, &dot));
        a = r1 / dot;

        cublasErrchk(hipblasDaxpy(cublasHandle, matrice_size, &a, p_d, 1, x_d, 1));
        na = -a;
        cublasErrchk(hipblasDaxpy(cublasHandle, matrice_size, &na, Ax_d, 1, rhs_d, 1));

        r0 = r1;
        cublasErrchk(hipblasDdot(cublasHandle, matrice_size, rhs_d, 1, rhs_d, 1, &r1));
        cudaErrchk(hipStreamSynchronize(stream));
        if(flag_verbose){
            std::printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        }
        k++;
    }

    //end CG
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipStreamSynchronize(stream));
    time += omp_get_wtime();

    //copy solution to host
    if(flag_verbose){
        std::printf("Copy solution to host\n");
    }
    cudaErrchk(hipMemcpy(rhs_h, x_d, matrice_size * sizeof(double), hipMemcpyDeviceToHost));


    if(!assert_same_array<double>(rhs_h, reference_solution_h, tolerance, matrice_size)){
        std::printf("Error: CG solution is not the same as the reference solution\n");
        return -1.0;
    }
    else{
        std::printf("CG solution is the same as the reference solution\n");
    }


    if (buffer) {
        cudaErrchk(hipFree(buffer));
    }
    if(cusparseHandle) {
        cusparseErrchk(hipsparseDestroy(cusparseHandle));
    }
    if(cublasHandle) {
        cublasErrchk(hipblasDestroy(cublasHandle));
    }
    if(stream) {
        cudaErrchk(hipStreamDestroy(stream));
    }
    if(matA) {
        cusparseErrchk(hipsparseDestroySpMat(matA));
    }
    if(vecx) {
        cusparseErrchk(hipsparseDestroyDnVec(vecx));
    }
    if(vecAx) {
        cusparseErrchk(hipsparseDestroyDnVec(vecAx));
    }
    if(vecp) {
        cusparseErrchk(hipsparseDestroyDnVec(vecp));
    }

    if(data_d){
        cudaErrchk(hipFree(data_d));
    }
    if(col_indices_d){
        cudaErrchk(hipFree(col_indices_d));
    }
    if(row_indptr_d){
        cudaErrchk(hipFree(row_indptr_d));
    }
    if(rhs_d){
        cudaErrchk(hipFree(rhs_d));
    }
    if(x_d){
        cudaErrchk(hipFree(x_d));
    }
    if(p_d){
        cudaErrchk(hipFree(p_d));
    }
    if(Ax_d){
        cudaErrchk(hipFree(Ax_d));
    }

    return time;
}


