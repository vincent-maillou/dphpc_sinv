#include "hip/hip_runtime.h"
#include <string> 
#include <omp.h>

#include "utils.h"

#include "mkl.h"
#include "hipsolver.h"
#include <hipsparse.h>



// cusolver has HIPSOLVER_STATUS_SUCCESS and not hipSuccess, but they are the same
// this seems again kinda hacky
#define cudaErrchk(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDAassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define cusolverErrchk(ans) { cusolverAssert((ans), __FILE__, __LINE__); }
inline void cusolverAssert(hipsolverStatus_t code, const char *file, int line, bool abort=true)
{
   if (code != HIPSOLVER_STATUS_SUCCESS) 
   {
        //Did not find a counter part to hipGetErrorString in cusolver
        fprintf(stderr,"CUSOLVERassert: %s %d\n", file, line);
        if (abort) exit(code);
   }
}


#define cublasErrchk(ans) { cublasAssert((ans), __FILE__, __LINE__); }
inline void cublasAssert(hipblasStatus_t code, const char *file, int line, bool abort=true)
{
   if (code != HIPBLAS_STATUS_SUCCESS) 
   {
        //Did not find a counter part to hipGetErrorString in cusolver
        fprintf(stderr,"CUBLASassert: %s %d\n", file, line);
        if (abort) exit(code);
   }
}

#define cusparseErrchk(ans) { cusparseAssert((ans), __FILE__, __LINE__); }
inline void cusparseAssert(hipsparseStatus_t code, const char *file, int line, bool abort=true)
{
   if (code != HIPSPARSE_STATUS_SUCCESS) 
   {
        //Did not find a counter part to hipGetErrorString in cusolver
        fprintf(stderr,"CUSPARSEassert: %s %d\n", file, line);
        if (abort) exit(code);
   }
}

hipsolverHandle_t CreateCusolverDnHandle(int device) {
    if (hipSetDevice(device) != hipSuccess) {
        throw std::runtime_error("Failed to set CUDA device.");
    }
    hipsolverHandle_t handle;
    cusolverErrchk(hipsolverDnCreate(&handle));
    return handle;
}


double solve_mkl_dgesv(
    double *matrix_dense,
    double *rhs,
    double *reference_solution,
    int matrix_size,
    double tolerance,
    bool flag_verbose)
{

    double time = -1.0;


    int ipiv[matrix_size];
    int nrhs = 1;
    int info;
    time = -omp_get_wtime();
    dgesv(&matrix_size, &nrhs, matrix_dense, &matrix_size, ipiv, rhs, &matrix_size, &info);
    time += omp_get_wtime();

    if(info != 0){
        std::printf("Error in MKL dgesv\n");
        std::printf("info: %d\n", info);
        return -1.0;
    }

    if(flag_verbose){
        std::printf("MKL dgesv done\n");
    }

    if(!assert_same_array<double>(rhs, reference_solution, tolerance, matrix_size)){
        std::printf("Error: MKL dgesv solution is not the same as the reference solution\n");
        return -1.0;
    }
    else{
        std::printf("MKL dgesv solution is the same as the reference solution\n");
    }
    return time;
}

double solve_cusolver_LU(
    double *matrix_dense_h,
    double *rhs_h,
    double *reference_solution_h,
    int matrix_size,
    double tolerance,
    bool flag_verbose)
{

    double time = -1.0;
    hipStream_t stream = NULL;
    hipsolverHandle_t handle = CreateCusolverDnHandle(0);
    cudaErrchk(hipStreamCreate(&stream));
    cusolverErrchk(hipsolverSetStream(handle, stream));



    int info_h = 0;
    int bufferSize = 0;

    double *matrix_dense_d = NULL;
    double *rhs_d = NULL;
    int *ipiv_d = NULL;
    int *info_d = NULL;
    double *buffer = NULL;

    //allocate memory on device
    cudaErrchk(hipMalloc((void**)&info_d, sizeof(int)))
    cudaErrchk(hipMalloc((void**)&matrix_dense_d, matrix_size*matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&rhs_d, matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&ipiv_d, matrix_size*sizeof(int)));


    //copy data to device
    if(flag_verbose){
        std::printf("Copy data to device\n");
    }
    cudaErrchk(hipMemcpy(matrix_dense_d, matrix_dense_h, matrix_size*matrix_size*sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemset(info_d, 0, sizeof(int)));
    cudaErrchk(hipMemcpy(rhs_d, rhs_h, matrix_size*sizeof(double), hipMemcpyHostToDevice));


    //figure out extra amount of memory needed
    cusolverErrchk(hipsolverDnDgetrf_bufferSize(handle, matrix_size, matrix_size,
                                            (double *)matrix_dense_d,
                                              matrix_size, &bufferSize));
    cudaErrchk(hipMalloc(&buffer, sizeof(double) * bufferSize));

    //LU factorization
    if(flag_verbose){
        std::printf("LU factorization\n");
    }
    time = -omp_get_wtime();
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipStreamSynchronize(stream));
    cusolverErrchk(hipsolverDnDgetrf(handle, matrix_size, matrix_size,
                                matrix_dense_d, matrix_size, buffer, ipiv_d, info_d));
    
    //copy info to host
    cudaErrchk(hipMemcpy(&info_h, info_d, sizeof(int), hipMemcpyDeviceToHost));

    if (info_h != 0) {
        fprintf(stderr, "Error: LU factorization failed\n");
    }
    else{
        std::printf("LU factorization done\n");
    }

    if(flag_verbose){
        std::printf("Back substitution\n");
    }
    //back substitution
    cusolverErrchk(hipsolverDnDgetrs(handle, HIPBLAS_OP_N, matrix_size,
                                    1, matrix_dense_d, matrix_size, ipiv_d,
                                    rhs_d, matrix_size, info_d));
    cudaErrchk(hipStreamSynchronize(stream));
    cudaErrchk(hipDeviceSynchronize());
    time += omp_get_wtime();


    cudaErrchk(hipMemcpy(&info_h, info_d, sizeof(int), hipMemcpyDeviceToHost));
    if (info_h != 0) {
        fprintf(stderr, "Error: Back substitution failed\n");
    }
    else{
        std::printf("Back substitution done\n");
    }

    //copy solution to host
    if(flag_verbose){
        std::printf("Copy solution to host\n");
    }
    cudaErrchk(hipMemcpy(rhs_h, rhs_d, matrix_size*sizeof(double), hipMemcpyDeviceToHost));

    if(!assert_same_array<double>(rhs_h, reference_solution_h, tolerance, matrix_size)){
        std::printf("Error: CuSolver solution is not the same as the reference solution\n");
        return -1.0;
    }
    else{
        std::printf("CuSolver solution is the same as the reference solution\n");
    }


    if (info_d) {
        cudaErrchk(hipFree(info_d));
    }
    if (buffer) {
        cudaErrchk(hipFree(buffer));
    }
    if (matrix_dense_d) {
        cudaErrchk(hipFree(matrix_dense_d));
    }
    if (ipiv_d) {
        cudaErrchk(hipFree(ipiv_d));
    }


    if (handle) {
        cusolverErrchk(hipsolverDnDestroy(handle));
    }
    if (stream) {
        cudaErrchk(hipStreamDestroy(stream));
    }

    return time;
}


double solve_cusparse_CG(
    double *data_h,
    int *col_indices_h,
    int *row_indptr_h,
    double *rhs_h,
    double *reference_solution_h,
    int nnz,
    int matrix_size,
    double tolerance,
    bool flag_verbose){

    double time = -1.0;
    hipStream_t stream = NULL;
    
    
    hipsparseHandle_t cusparseHandle = 0;
    cusparseErrchk(hipsparseCreate(&cusparseHandle));    

    hipblasHandle_t cublasHandle = 0;
    cublasErrchk(hipblasCreate(&cublasHandle));

    cudaErrchk(hipStreamCreate(&stream));
    cusparseErrchk(hipsparseSetStream(cusparseHandle, stream));
    cublasErrchk(hipblasSetStream(cublasHandle, stream));


    double *data_d = NULL;
    double *col_indices_d = NULL;
    double *row_indptr_d = NULL;
    double *rhs_d = NULL;
    double *x_d = NULL;
    double *p_d = NULL;
    double *Ax_d = NULL;
    double dot;

    hipsparseSpMatDescr_t matA = NULL;

    const double tol = 1.e-15;
    const int max_iter = 2000;
    double a, b, na;
    double alpha, beta, alpham1, r0, r1;
    size_t bufferSize = 0;
    void *buffer = NULL;

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.0;


    //allocate memory on device
    cudaErrchk(hipMalloc((void**)&data_d, nnz*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&col_indices_d, nnz*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&row_indptr_d, (matrix_size+1)*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&rhs_d, matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&x_d, matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void **)&p_d, matrix_size * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&Ax_d, matrix_size * sizeof(double)));

    /* Wrap raw data into cuSPARSE generic API objects */
    cusparseErrchk(hipsparseCreateCsr(&matA, matrix_size, matrix_size,
                                        nnz, row_indptr_d, col_indices_d, data_d,
                                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));



    hipsparseDnVecDescr_t vecx = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecx, matrix_size, x_d, HIP_R_64F));
    hipsparseDnVecDescr_t vecp = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecp, matrix_size, p_d, HIP_R_64F));
    hipsparseDnVecDescr_t vecAx = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecAx, matrix_size, Ax_d, HIP_R_64F));


    //copy data to device
    if(flag_verbose){
        std::printf("Copy data to device\n");
    }
    cudaErrchk(hipMemcpy(rhs_d, rhs_h, matrix_size*sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(col_indices_d, col_indices_h, nnz * sizeof(int), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(row_indptr_d, row_indptr_h, (matrix_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(data_d, data_h, nnz * sizeof(double), hipMemcpyHostToDevice));
    
    // setting starting guess to zero
    cudaErrchk(hipMemset(x_d, 0.0, matrix_size*sizeof(double)))
    

    //figure out extra amount of memory needed
    if(flag_verbose){
        std::printf("Figure out extra amount of memory needed\n");
    }
    cusparseErrchk(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecx,
        &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    cudaErrchk(hipMalloc(&buffer, bufferSize));


    //begin CG
    time = -omp_get_wtime();
    cudaErrchk(hipStreamSynchronize(stream));
    cudaErrchk(hipDeviceSynchronize());
    if(flag_verbose){
        std::printf("CG starts\n");
    }

    // calc A*x
    cusparseErrchk(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, vecx, &beta, vecAx, HIP_R_64F,
                               HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

    // r = b - A*x
    cublasErrchk(hipblasDaxpy(cublasHandle, matrix_size, &alpham1, Ax_d, 1, rhs_d, 1));
    cublasErrchk(hipblasDdot(cublasHandle, matrix_size, rhs_d, 1, rhs_d, 1, &r1));


    int k = 1;
    while (r1 > tol * tol && k <= max_iter) {
        if(k > 1){
            b = r1 / r0;
            cublasErrchk(hipblasDscal(cublasHandle, matrix_size, &b, p_d, 1));
            cublasErrchk(hipblasDaxpy(cublasHandle, matrix_size, &alpha, rhs_d, 1, p_d, 1));            
        }
        else {
            cublasErrchk(hipblasDcopy(cublasHandle, matrix_size, rhs_d, 1, p_d, 1));
        }

        cusparseErrchk(hipsparseSpMV(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecp,
            &beta, vecAx, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
    
        cublasErrchk(hipblasDdot(cublasHandle, matrix_size, p_d, 1, Ax_d, 1, &dot));
        a = r1 / dot;

        cublasErrchk(hipblasDaxpy(cublasHandle, matrix_size, &a, p_d, 1, x_d, 1));
        na = -a;
        cublasErrchk(hipblasDaxpy(cublasHandle, matrix_size, &na, Ax_d, 1, rhs_d, 1));

        r0 = r1;
        cublasErrchk(hipblasDdot(cublasHandle, matrix_size, rhs_d, 1, rhs_d, 1, &r1));
        cudaErrchk(hipStreamSynchronize(stream));

        k++;
    }

    if(flag_verbose){
        std::printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
    }

    //end CG
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipStreamSynchronize(stream));
    time += omp_get_wtime();

    //copy solution to host
    if(flag_verbose){
        std::printf("Copy solution to host\n");
    }
    cudaErrchk(hipMemcpy(rhs_h, x_d, matrix_size * sizeof(double), hipMemcpyDeviceToHost));


    if(!assert_same_array<double>(rhs_h, reference_solution_h, tolerance, matrix_size)){
        std::printf("Error: CG solution is not the same as the reference solution\n");
        return -1.0;
    }
    else{
        std::printf("CG solution is the same as the reference solution\n");
    }


    if (buffer) {
        cudaErrchk(hipFree(buffer));
    }
    if(cusparseHandle) {
        cusparseErrchk(hipsparseDestroy(cusparseHandle));
    }
    if(cublasHandle) {
        cublasErrchk(hipblasDestroy(cublasHandle));
    }
    if(stream) {
        cudaErrchk(hipStreamDestroy(stream));
    }
    if(matA) {
        cusparseErrchk(hipsparseDestroySpMat(matA));
    }
    if(vecx) {
        cusparseErrchk(hipsparseDestroyDnVec(vecx));
    }
    if(vecAx) {
        cusparseErrchk(hipsparseDestroyDnVec(vecAx));
    }
    if(vecp) {
        cusparseErrchk(hipsparseDestroyDnVec(vecp));
    }

    if(data_d){
        cudaErrchk(hipFree(data_d));
    }
    if(col_indices_d){
        cudaErrchk(hipFree(col_indices_d));
    }
    if(row_indptr_d){
        cudaErrchk(hipFree(row_indptr_d));
    }
    if(rhs_d){
        cudaErrchk(hipFree(rhs_d));
    }
    if(x_d){
        cudaErrchk(hipFree(x_d));
    }
    if(p_d){
        cudaErrchk(hipFree(p_d));
    }
    if(Ax_d){
        cudaErrchk(hipFree(Ax_d));
    }

    return time;
}

double solve_cusparse_ILU_CG(
    double *data_h,
    int *col_indices_h,
    int *row_indptr_h,
    double *rhs_h,
    double *reference_solution_h,
    int nnz,
    int matrix_size,
    double tolerance,
    bool flag_verbose)
{

    double time = -1.0;

    
    
    hipsparseHandle_t cusparseHandle = 0;
    cusparseErrchk(hipsparseCreate(&cusparseHandle));    

    hipblasHandle_t cublasHandle = 0;
    cublasErrchk(hipblasCreate(&cublasHandle));

    hipStream_t stream = NULL;
    cudaErrchk(hipStreamCreate(&stream));

    cusparseErrchk(hipsparseSetStream(cusparseHandle, stream));
    cublasErrchk(hipblasSetStream(cublasHandle, stream));


    double *data_d = NULL;
    int *col_indices_d = NULL;
    int *row_indptr_d = NULL;
    double *rhs_d = NULL;
    double *x_d = NULL;
    double *p_d = NULL;
    double *Ax_d = NULL;
    double *valsILU0_d = NULL;
    double *zm1_d = NULL;
    double *zm2_d = NULL;
    double *rm2_d = NULL;
    double *omega_d = NULL;
    double *y_d = NULL;

    const double tol = 1.e-15;
    const int max_iter = 2000;
    double alpha, beta, r1;
    double numerator, denominator, nalpha;
    const double doubleone = 1.0;
    const double doublezero = 0.0;

    alpha = 1.0;
    beta = 0.0;


    hipsparseSpMatDescr_t matA = NULL;
    hipsparseSpMatDescr_t matM_lower = NULL;
    hipsparseSpMatDescr_t matM_upper = NULL;
    hipsparseFillMode_t   fill_lower    = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseDiagType_t   diag_unit     = HIPSPARSE_DIAG_TYPE_UNIT;
    hipsparseFillMode_t   fill_upper    = HIPSPARSE_FILL_MODE_UPPER;
    hipsparseDiagType_t   diag_non_unit = HIPSPARSE_DIAG_TYPE_NON_UNIT;


    int                 bufferSizeLU = 0;
    size_t              bufferSizeMV, bufferSizeL, bufferSizeU;
    void*               bufferLU_d, *bufferMV_d,  *bufferL_d, *bufferU_d;
    hipsparseSpSVDescr_t spsvDescrL, spsvDescrU;
    hipsparseMatDescr_t   matLU;
    csrilu02Info_t      infoILU = NULL;


    /* Description of the A matrix */
    hipsparseMatDescr_t descr = 0;
    cusparseErrchk(hipsparseCreateMatDescr(&descr));
    cusparseErrchk(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseErrchk(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

    //allocate memory on device
    cudaErrchk(hipMalloc((void**)&data_d, nnz*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&col_indices_d, nnz*sizeof(int)));
    cudaErrchk(hipMalloc((void**)&row_indptr_d, (matrix_size+1)*sizeof(int)));
    cudaErrchk(hipMalloc((void**)&rhs_d, matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void**)&x_d, matrix_size*sizeof(double)));
    cudaErrchk(hipMalloc((void **)&y_d, matrix_size * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&p_d, matrix_size * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&Ax_d, matrix_size * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&omega_d, matrix_size * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&valsILU0_d, nnz * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&zm1_d, (matrix_size) * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&zm2_d, (matrix_size) * sizeof(double)));
    cudaErrchk(hipMalloc((void **)&rm2_d, (matrix_size) * sizeof(double)));


    /* Wrap raw data into cuSPARSE generic API objects */
    hipsparseDnVecDescr_t vecp = NULL, vecX=NULL, vecY = NULL, vecR = NULL, vecZM1=NULL;
    hipsparseDnVecDescr_t vecomega = NULL;
    cusparseErrchk(hipsparseCreateDnVec(&vecp, matrix_size, p_d, HIP_R_64F));
    cusparseErrchk(hipsparseCreateDnVec(&vecX, matrix_size, x_d, HIP_R_64F));
    cusparseErrchk(hipsparseCreateDnVec(&vecY, matrix_size, y_d, HIP_R_64F));
    cusparseErrchk(hipsparseCreateDnVec(&vecR, matrix_size, rhs_d, HIP_R_64F));
    cusparseErrchk(hipsparseCreateDnVec(&vecZM1, matrix_size, zm1_d, HIP_R_64F));
    cusparseErrchk(hipsparseCreateDnVec(&vecomega, matrix_size, omega_d, HIP_R_64F));


    //copy data to device
    if(flag_verbose){
        std::printf("Copy data to device\n");
    }
    cudaErrchk(hipMemcpy(rhs_d, rhs_h, matrix_size*sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(col_indices_d, col_indices_h, nnz * sizeof(int), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(row_indptr_d, row_indptr_h, (matrix_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(data_d, data_h, nnz * sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(valsILU0_d, data_d, nnz*sizeof(double), hipMemcpyDeviceToDevice));
    // setting starting guess to zero
    cudaErrchk(hipMemset(x_d, 0.0, matrix_size*sizeof(double)))


    cusparseErrchk(hipsparseCreateCsr(
        &matA, matrix_size, matrix_size, nnz, row_indptr_d, col_indices_d, data_d, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    
    //Lower Part 
     cusparseErrchk(hipsparseCreateCsr(&matM_lower, matrix_size, matrix_size, nnz, row_indptr_d, col_indices_d, valsILU0_d,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    cusparseErrchk(hipsparseSpMatSetAttribute(matM_lower,
                                              HIPSPARSE_SPMAT_FILL_MODE,
                                              &fill_lower, sizeof(fill_lower)));
    cusparseErrchk(hipsparseSpMatSetAttribute(matM_lower,
                                              HIPSPARSE_SPMAT_DIAG_TYPE,
                                              &diag_unit, sizeof(diag_unit)));
    // M_upper
    cusparseErrchk(hipsparseCreateCsr(&matM_upper, matrix_size, matrix_size, nnz, row_indptr_d, col_indices_d, valsILU0_d,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    cusparseErrchk(hipsparseSpMatSetAttribute(matM_upper,
                                              HIPSPARSE_SPMAT_FILL_MODE,
                                              &fill_upper, sizeof(fill_upper)));
    cusparseErrchk(hipsparseSpMatSetAttribute(matM_upper,
                                              HIPSPARSE_SPMAT_DIAG_TYPE,
                                              &diag_non_unit,
                                              sizeof(diag_non_unit)));


    /* Create ILU(0) info object */
    cusparseErrchk(hipsparseCreateCsrilu02Info(&infoILU));
    cusparseErrchk(hipsparseCreateMatDescr(&matLU) );
    cusparseErrchk(hipsparseSetMatType(matLU, HIPSPARSE_MATRIX_TYPE_GENERAL) );
    cusparseErrchk(hipsparseSetMatIndexBase(matLU, HIPSPARSE_INDEX_BASE_ZERO) );

    /* Allocate workspace for cuSPARSE */
    if(flag_verbose){
        std::printf("Figure out extra amount of memory needed\n");
    }
    cusparseErrchk(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone, matA,
        vecp, &doublezero, vecomega, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
        &bufferSizeMV));
    cudaErrchk( hipMalloc(&bufferMV_d, bufferSizeMV) );

    cusparseErrchk(hipsparseDcsrilu02_bufferSize(
        cusparseHandle, matrix_size, nnz, matLU, data_d, row_indptr_d, col_indices_d, infoILU, &bufferSizeLU));
    cudaErrchk( hipMalloc(&bufferLU_d, bufferSizeLU) );

    cusparseErrchk(hipsparseSpSV_createDescr(&spsvDescrL) );
    cusparseErrchk(hipsparseSpSV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone, matM_lower, vecR, vecX, HIP_R_64F,
        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &bufferSizeL));
    cudaErrchk(hipMalloc(&bufferL_d, bufferSizeL) );

    cusparseErrchk(hipsparseSpSV_createDescr(&spsvDescrU) );
    cusparseErrchk(hipsparseSpSV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone, matM_upper, vecR, vecX, HIP_R_64F,
        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, &bufferSizeU));
    cudaErrchk(hipMalloc(&bufferU_d, bufferSizeU) );



    //begin CG
    time = -omp_get_wtime();
    cudaErrchk(hipStreamSynchronize(stream));
    cudaErrchk(hipDeviceSynchronize());
    if(flag_verbose){
        std::printf("CG starts\n");
    }

    /* Preconditioned Conjugate Gradient using ILU.
       --------------------------------------------
       Follows the description by Golub & Van Loan,
       "Matrix Computations 3rd ed.", Algorithm 10.3.1  */

    printf("\nConvergence of CG using ILU(0) preconditioning: \n");



    /* Perform analysis for ILU(0) */
    cusparseErrchk(hipsparseDcsrilu02_analysis(
        cusparseHandle, matrix_size, nnz, descr, valsILU0_d, row_indptr_d, col_indices_d, infoILU,
        HIPSPARSE_SOLVE_POLICY_USE_LEVEL, bufferLU_d));

    /* generate the ILU(0) factors */
    cusparseErrchk(hipsparseDcsrilu02(
        cusparseHandle, matrix_size, nnz, matLU, valsILU0_d, row_indptr_d, col_indices_d, infoILU,
        HIPSPARSE_SOLVE_POLICY_USE_LEVEL, bufferLU_d));

    /* perform triangular solve analysis */
    cusparseErrchk(hipsparseSpSV_analysis(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone,
        matM_lower, vecR, vecX, HIP_R_64F,
        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, bufferL_d));

    cusparseErrchk(hipsparseSpSV_analysis(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone,
        matM_upper, vecR, vecX, HIP_R_64F,
        HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, bufferU_d));

    // /* reset the initial guess of the solution to zero */
    // for (int i = 0; i < matrix_size; i++)
    // {
    //     x[i] = 0.0;
    // }
    // cudaErrchk(hipMemcpy(
    //     rhs_d, rhs, matrix_size * sizeof(double), hipMemcpyHostToDevice));
    // cudaErrchk(hipMemcpy(
    //     x_d, x, matrix_size * sizeof(double), hipMemcpyHostToDevice));

    int k = 0;
    cublasErrchk(hipblasDdot(cublasHandle, matrix_size, rhs_d, 1, rhs_d, 1, &r1));

    while (r1 > tol * tol && k <= max_iter)
    {
        // preconditioner application: zm1_d = U^-1 L^-1 rhs_d
        cusparseErrchk(hipsparseSpSV_solve(cusparseHandle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone,
            matM_lower, vecR, vecY, HIP_R_64F,
            HIPSPARSE_SPSV_ALG_DEFAULT,
            spsvDescrL) );
            
        cusparseErrchk(hipsparseSpSV_solve(cusparseHandle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone, matM_upper,
            vecY, vecZM1,
            HIP_R_64F,
            HIPSPARSE_SPSV_ALG_DEFAULT,
            spsvDescrU));
        k++;

        if (k == 1)
        {
            cublasErrchk(hipblasDcopy(cublasHandle, matrix_size, zm1_d, 1, p_d, 1));
        }
        else
        {
            cublasErrchk(hipblasDdot(
                cublasHandle, matrix_size, rhs_d, 1, zm1_d, 1, &numerator));
            cublasErrchk(hipblasDdot(
                cublasHandle, matrix_size, rm2_d, 1, zm2_d, 1, &denominator));
            beta = numerator / denominator;
            cublasErrchk(hipblasDscal(cublasHandle, matrix_size, &beta, p_d, 1));
            cublasErrchk(hipblasDaxpy(
                cublasHandle, matrix_size, &doubleone, zm1_d, 1, p_d, 1));
        }

        cusparseErrchk(hipsparseSpMV(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &doubleone, matA,
            vecp, &doublezero, vecomega, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
            bufferMV_d));
    
        cublasErrchk(hipblasDdot(
            cublasHandle, matrix_size, rhs_d, 1, zm1_d, 1, &numerator));
        cublasErrchk(hipblasDdot(
            cublasHandle, matrix_size, p_d, 1, omega_d, 1, &denominator));

        alpha = numerator / denominator;
        cublasErrchk(hipblasDaxpy(cublasHandle, matrix_size, &alpha, p_d, 1, x_d, 1));
        cublasErrchk(hipblasDcopy(cublasHandle, matrix_size, rhs_d, 1, rm2_d, 1));
        cublasErrchk(hipblasDcopy(cublasHandle, matrix_size, zm1_d, 1, zm2_d, 1));
        nalpha = -alpha;
        cublasErrchk(hipblasDaxpy(
            cublasHandle, matrix_size, &nalpha, omega_d, 1, rhs_d, 1));
        cublasErrchk(hipblasDdot(cublasHandle, matrix_size, rhs_d, 1, rhs_d, 1, &r1));
    }

    printf("  iteration = %3d, residual = %e \n", k, sqrt(r1));


    //end CG
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipStreamSynchronize(stream));
    time += omp_get_wtime();

    //copy solution to host
    if(flag_verbose){
        std::printf("Copy solution to host\n");
    }
    cudaErrchk(hipMemcpy(rhs_h, x_d, matrix_size * sizeof(double), hipMemcpyDeviceToHost));


    if(!assert_same_array<double>(rhs_h, reference_solution_h, tolerance, matrix_size)){
        std::printf("Error: ILU CG solution is not the same as the reference solution\n");
        return -1.0;
    }
    else{
        std::printf("ILU CG solution is the same as the reference solution\n");
    }


    /* Destroy descriptors */
    if(descr) {
        cusparseErrchk(hipsparseDestroyMatDescr(descr));
    }
    if(matA) {
        cusparseErrchk(hipsparseDestroySpMat(matA));
    }
    if(vecp) {
        cusparseErrchk(hipsparseDestroyDnVec(vecp));
    }
    if(vecX) {
        cusparseErrchk(hipsparseDestroyDnVec(vecX));
    }
    if(vecY) {
        cusparseErrchk(hipsparseDestroyDnVec(vecY));
    }
    if(vecR) {
        cusparseErrchk(hipsparseDestroyDnVec(vecR));
    }
    if(vecZM1) {
        cusparseErrchk(hipsparseDestroyDnVec(vecZM1));
    }
    if(vecomega) {
        cusparseErrchk(hipsparseDestroyDnVec(vecomega));
    }
    if(matM_lower) {
        cusparseErrchk(hipsparseDestroySpMat(matM_lower));
    }
    if(matM_upper) {
        cusparseErrchk(hipsparseDestroySpMat(matM_upper));
    }
    if(matLU) {
        cusparseErrchk(hipsparseDestroyMatDescr(matLU));
    }
    if(spsvDescrL) {
        cusparseErrchk(hipsparseSpSV_destroyDescr(spsvDescrL));
    }
    if(spsvDescrU) {
        cusparseErrchk(hipsparseSpSV_destroyDescr(spsvDescrU));
    }
    if(infoILU) {
        cusparseErrchk(hipsparseDestroyCsrilu02Info(infoILU));
    }


    //Destroy handles
    if(cusparseHandle) {
        cusparseErrchk(hipsparseDestroy(cusparseHandle));
    }
    if(cublasHandle) {
        cublasErrchk(hipblasDestroy(cublasHandle));
    }
    if(stream) {
        cudaErrchk(hipStreamDestroy(stream));
    }


    // Destroy buffer
    //bufferLU_d, *bufferMV_d,  *bufferL_d, *bufferU_d;
    if (bufferLU_d) {
        cudaErrchk(hipFree(bufferLU_d));
    }
    if (bufferMV_d) {
        cudaErrchk(hipFree(bufferMV_d));
    }
    if (bufferL_d) {
        cudaErrchk(hipFree(bufferL_d));
    }
    if (bufferU_d) {
        cudaErrchk(hipFree(bufferU_d));
    }


    if(data_d){
        cudaErrchk(hipFree(data_d));
    }
    if(col_indices_d){
        cudaErrchk(hipFree(col_indices_d));
    }
    if(row_indptr_d){
        cudaErrchk(hipFree(row_indptr_d));
    }
    if(rhs_d){
        cudaErrchk(hipFree(rhs_d));
    }
    if(x_d){
        cudaErrchk(hipFree(x_d));
    }
    if(p_d){
        cudaErrchk(hipFree(p_d));
    }
    if(Ax_d){
        cudaErrchk(hipFree(Ax_d));
    }
    if(valsILU0_d){
        cudaErrchk(hipFree(valsILU0_d));
    }
    if(zm1_d){
        cudaErrchk(hipFree(zm1_d));
    }
    if(zm2_d){
        cudaErrchk(hipFree(zm2_d));
    }
    if(rm2_d){
        cudaErrchk(hipFree(rm2_d));
    }

    return time;
}


